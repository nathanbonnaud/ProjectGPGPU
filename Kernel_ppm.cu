#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "ppm_lib.h"

#define divisionFactor 9
#define N 500*1000
static void HandleError( hipError_t err, const char *file, int line ) {
if (err != hipSuccess) {
printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
exit( EXIT_FAILURE ); }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define CREATOR "PARALLELISME2OPENMP"



PPMImage *readPPM(const char *filename)
{
         char buff[16];
         PPMImage *img;
         FILE *fp;
         int c, rgb_comp_color;
         fp = fopen(filename, "rb");
         if (!fp) {
              fprintf(stderr, "Unable to open file '%s'\n", filename);
              exit(1);
         }

         if (!fgets(buff, sizeof(buff), fp)) {
              perror(filename);
              exit(1);
         }

    if (buff[0] != 'P' || buff[1] != '6') {
         fprintf(stderr, "Invalid image format (must be 'P6')\n");
         exit(1);
    }

    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    c = getc(fp);
    while (c == '#') {
    while (getc(fp) != '\n') ;
         c = getc(fp);
    }

    ungetc(c, fp);
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
         fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
         exit(1);
    }

    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
         fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
         exit(1);
    }

    if (rgb_comp_color!= RGB_COMPONENT_COLOR) {
         fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
         exit(1);
    }

    while (fgetc(fp) != '\n') ;
    img->data = (PPMPixel*)malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    if (fread(img->data, sizeof(PPMPixel)*img->x, img->y, fp) != img->y) {
         fprintf(stderr, "Error loading image '%s'\n", filename);
         exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(const char *filename, PPMImage *img)
{
    FILE *fp;
    fp = fopen(filename, "wb");
    if (!fp) {
         fprintf(stderr, "Unable to open file '%s'\n", filename);
         exit(1);
    }

    fprintf(fp, "P6\n");
    fprintf(fp, "# Created by %s\n",CREATOR);
    fprintf(fp, "%d %d\n",img->x,img->y);

    fprintf(fp, "%d\n", RGB_COMPONENT_COLOR);

    fwrite(img->data, 3 * img->x, img->y, fp);
    fclose(fp);
}




// GPU version 1//
// un bloc par pixel et un bloc par ligne on incrémente//  
__global__ void filterSofter(PPMPixel *img,int *filter ,PPMPixel *destination){
     
         
         int  finalRed =0;
         int  finalGreen  =0;
         int  finalBlue =0;
         int indFiltre = 0;
         
         int tidX =threadIdx.x+ blockIdx.x*blockDim.x;
         int l=tidX/500;          
         int c=tidX%500;
         int ll;
         int cc;

        for(int i=-2;i<=2;i++){
            for(int j=-2;j<=2;j++){
            ll=l+i;
            cc=c+j;

            if(ll<0 ){
                ll=-ll;
            }else if(ll>1000){
              ll=l-i;
            }
            if(cc<0 ){
                cc=-cc;
            } else if (cc>500){
              cc=c-j;
            }

            finalRed += img[(ll)*500+(cc)].red * filter[indFiltre];  
            finalGreen +=  img[(ll)*500+(cc)].green * filter[indFiltre]; 
            finalBlue +=  img[(ll)*500+(cc)].blue * filter[indFiltre]; 
            indFiltre++;
        
            }
        }
        
        
         destination[tidX].red =  finalRed / divisionFactor;
         destination[tidX].green = finalGreen / divisionFactor;
         destination[tidX].blue =  finalBlue / divisionFactor;
        
    
}



int main(){

    PPMImage *image, *imageCopy;
    image = readPPM("imageProject.ppm");
    imageCopy = readPPM("imageProject.ppm");
    

int filter[25] = { 1,   2,   0,   -2,   -1,
                           4 ,  8,   0 ,  -8 ,  -4,
                           6  , 12 , 0 ,  -12  , -6 ,
                           4,   8,   0 ,  -8,    -4,
                           1,   2,   0,   -2,   -1 };


PPMPixel *dev_image;
PPMPixel*dev_imageCopy;
int *dev_filter;

//double time;
//hipEvent_t start,stop;


HANDLE_ERROR( hipMalloc( (void**)&dev_image, image->x*image->y *3* sizeof(char) ) );
HANDLE_ERROR( hipMalloc( (void**)&dev_imageCopy, imageCopy->x*imageCopy->y*3 * sizeof(char) ) );
HANDLE_ERROR( hipMalloc( (void**)&dev_filter, 25 * sizeof(int) ));


/* copier 'a' et 'b' sur le GPU */

HANDLE_ERROR( hipMemcpy( dev_image, image->data,image->x*image->y *3* sizeof(char),hipMemcpyHostToDevice));
HANDLE_ERROR( hipMemcpy( dev_imageCopy, imageCopy->data, imageCopy->x*imageCopy->y *3* sizeof(char),hipMemcpyHostToDevice));
HANDLE_ERROR( hipMemcpy( dev_filter, filter, 25 * sizeof(int),hipMemcpyHostToDevice));


/*hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start, 0);
*/
filterSofter<<<500,1000>>>(dev_image,dev_filter,dev_imageCopy);


printf(">%s\n",hipGetErrorString (hipGetLastError ()));
/*
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&time, start, stop);
*/

/* copier le tableau 'c' depuis le GPU vers le CPU */
hipMemcpy( imageCopy->data, dev_imageCopy, imageCopy->x*imageCopy->y * 3*sizeof(char), hipMemcpyDeviceToHost);


//printf("Temps nécessaire :  %3.1f ms\n", time);

writePPM("imageProjectResult.ppm",imageCopy);

/* liberer la memoire allouee sur le GPU */
hipFree( dev_image );
hipFree( dev_imageCopy );
hipFree( dev_filter );

return 0;
}

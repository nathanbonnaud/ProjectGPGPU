#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "ppm_lib.h"

#define divisionFactor 9
static void HandleError( hipError_t err, const char *file, int line ) {
if (err != hipSuccess) {
printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
exit( EXIT_FAILURE ); }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define CREATOR "PARALLELISME2OPENMP"

struct filterCoeff{
 int l,c;
};

PPMImage *readPPM(const char *filename)
{
         char buff[16];
         PPMImage *img;
         FILE *fp;
         int c, rgb_comp_color;
         fp = fopen(filename, "rb");
         if (!fp) {
              fprintf(stderr, "Unable to open file '%s'\n", filename);
              exit(1);
         }

         if (!fgets(buff, sizeof(buff), fp)) {
              perror(filename);
              exit(1);
         }

    if (buff[0] != 'P' || buff[1] != '6') {
         fprintf(stderr, "Invalid image format (must be 'P6')\n");
         exit(1);
    }

    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    c = getc(fp);
    while (c == '#') {
    while (getc(fp) != '\n') ;
         c = getc(fp);
    }

    ungetc(c, fp);
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
         fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
         exit(1);
    }

    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
         fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
         exit(1);
    }

    if (rgb_comp_color!= RGB_COMPONENT_COLOR) {
         fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
         exit(1);
    }

    while (fgetc(fp) != '\n') ;
    img->data = (PPMPixel*)malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    if (fread(img->data, sizeof(PPMPixel)*img->x, img->y, fp) != img->y) {
         fprintf(stderr, "Error loading image '%s'\n", filename);
         exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(const char *filename, PPMImage *img)
{
    FILE *fp;
    fp = fopen(filename, "wb");
    if (!fp) {
         fprintf(stderr, "Unable to open file '%s'\n", filename);
         exit(1);
    }

    fprintf(fp, "P6\n");
    fprintf(fp, "# Created by %s\n",CREATOR);
    fprintf(fp, "%d %d\n",img->x,img->y);

    fprintf(fp, "%d\n", RGB_COMPONENT_COLOR);

    fwrite(img->data, 3 * img->x, img->y, fp);
    fclose(fp);
}





__global__ void filterSofter(PPMPixel *img,int* filter ,PPMPixel *destination, filterCoeff* coeff ){
     
         
          __shared__ int  finalRed;
          __shared__ int  finalGreen ;
          __shared__ int  finalBlue ;

        int tid = threadIdx.x;
        int tidX =threadIdx.x+ blockIdx.x*blockDim.x;
        int l=tidX/500;
        int c=tidX%500;
            
              if(tid==0){
              finalRed=0;
              finalGreen=0 ;
              finalBlue =0;
              }            
                    
               if( (c+coeff[tid].c + (l+coeff[tid].l)*500 )>=0&&(c+coeff[tid].c + (l+coeff[tid].l)*500 )<500*1000 ){

               finalRed+=img[c+coeff[tid].c + (l+coeff[tid].l)*500 ].red * filter[tid];
     //  printf("%d\n",finalRed);          
                finalGreen+=img[c+coeff[tid].c + (l+coeff[tid].l)*500 ].green * filter[tid];
              finalBlue +=img[c+coeff[tid].c + (l+coeff[tid].l)*500 ].blue * filter[tid];
          }

          __syncthreads();
          destination[tidX].red =  finalRed/divisionFactor;
         destination[tidX].green = finalGreen/divisionFactor;
         destination[tidX].blue =  finalBlue/divisionFactor;
    
}




int main(){

    PPMImage *image, *imageCopy;
    image = readPPM("imageProject.ppm");
    imageCopy = readPPM("imageProject.ppm");
    

int filter[25] = { 0,  0,   0,   0,   0,
                   0 ,  0 ,  0 ,  0 ,  0 ,
                   1 ,  2 ,  3 ,  2 ,  1 ,
                   0  , 0 ,  0 ,  0 ,  0,
                   0 ,  0 ,  0 ,  0 ,  0 };



 filterCoeff coeff[25] = {};

int k=0;
for(int i=-2;i<=2;i++)
    for(int j=-2;j<=2;j++)
            coeff[k++]={i,j};
 

PPMPixel *dev_image;
PPMPixel *dev_imageCopy;
int *dev_filter;
filterCoeff *dev_coeff;
//double time;
//hipEvent_t start,stop;


HANDLE_ERROR( hipMalloc( (void**)&dev_image, image->x*image->y *3* sizeof(char) ) );
HANDLE_ERROR( hipMalloc( (void**)&dev_imageCopy, imageCopy->x*imageCopy->y*3 * sizeof(char) ) );
HANDLE_ERROR( hipMalloc( (void**)&dev_filter, 25 * sizeof(int) ));
HANDLE_ERROR( hipMalloc( (void**)&dev_coeff, 25* sizeof( filterCoeff) ));

/* copier 'a' et 'b' sur le GPU */

HANDLE_ERROR( hipMemcpy( dev_image, image->data,image->x*image->y *3* sizeof(char),hipMemcpyHostToDevice));
HANDLE_ERROR( hipMemcpy( dev_imageCopy, imageCopy->data, imageCopy->x*imageCopy->y *3* sizeof(char),hipMemcpyHostToDevice));
HANDLE_ERROR( hipMemcpy( dev_filter, filter, 25 * sizeof(int),hipMemcpyHostToDevice));
HANDLE_ERROR( hipMemcpy( dev_coeff, coeff, 25 * sizeof( filterCoeff),hipMemcpyHostToDevice));

/*hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start, 0);
*/
filterSofter<<<20*1000,25>>>(dev_image,dev_filter,dev_imageCopy,dev_coeff);
hipDeviceSynchronize();

printf(">%s\n",hipGetErrorString (hipGetLastError ()));
/*
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&time, start, stop);
*/

/* copier le tableau 'c' depuis le GPU vers le CPU */
HANDLE_ERROR( hipMemcpy( imageCopy->data, dev_imageCopy, imageCopy->x*imageCopy->y * 3*sizeof(char), hipMemcpyDeviceToHost));


//printf("Temps nécessaire :  %3.1f ms\n", time);

writePPM("imageProjectResult.ppm",imageCopy);

/* liberer la memoire allouee sur le GPU */
hipFree( dev_image );
hipFree( dev_imageCopy );
hipFree( dev_filter );

return 0;
}
